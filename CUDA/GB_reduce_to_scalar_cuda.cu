
//------------------------------------------------------------------------------
// GB_reduce_to_scalar_cuda.cu: reduce on the GPU with semiring 
//------------------------------------------------------------------------------

// SPDX-License-Identifier: Apache-2.0
// SuiteSparse:GraphBLAS, Timothy A. Davis, (c) 2017-2019, All Rights Reserved.
// http://suitesparse.com   See GraphBLAS/Doc/License.txt for license.

//------------------------------------------------------------------------------

#include "GB_cuda.h"

#include "templates/reduceWarp.cu.jit"
#include "test/semiringFactory.hpp"
#include "jitify.hpp"

GrB_Info GB_reduce_to_scalar_cuda
(
    GB_void *s,
    const GrB_Monoid reduce,
    const GrB_Matrix A,
    GB_Context Contetxt
)
{ 

    printf ("Hi I am %s :-)\n", __FILE__) ;

    // result = sum (Anz [0..anz-1]) using the GPU,
    // with a kernel that has ntasks = grid.x and blocksize = blockDim.x
    // nthreads = # of GPUs to use, but 1 for now
    // We have a workspace W of size ntasks.

    thread_local static jitify::JitCache kernel_cache;

    // stringified kernel specified above
    jitify::Program program= kernel_cache.program( templates_reduceWarp_cu, 0, 0,
        file_callback_plus);
    //{"--use_fast_math", "-I/usr/local/cuda/include"});

    int nnz = GB_NNZ( A ) ;
    int blocksize = 1024 ;
    int ntasks = ( nnz + blocksize -1) / blocksize ;

    dim3 grid(ntasks);
    dim3 block(blocksize);

    using jitify::reflection::type_of;
    program.kernel("reduceWarp")
                    .instantiate(type_of(*Ax))
                    .configure(grid, block)
                    .launch(Ax, W, anz);

    hipDeviceSynchronize ( ) ;

    int64_t s = 0 ;
    for (int i = 0 ; i < ntasks ; i++)
    {
        s += W [i] ; 
    }

    (*result) = s ;

    return (GrB_SUCCESS) ;
}

