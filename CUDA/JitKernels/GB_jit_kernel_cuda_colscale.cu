#include "hip/hip_runtime.h"
using namespace cooperative_groups ;

// do not #include functions inside of other functions!
#include "GB_cuda_ek_slice.cuh"

#define log2_chunk_size 7
#define chunk_size 128

__global__ void GB_cuda_colscale_kernel
(
    GrB_Matrix C,
    GrB_Matirx A,
    GrB_Matrix D
)
{

    const GB_A_TYPE *__restrict__ Ax = (GB_A_TYPE *) A->x ;
    const GB_B_TYPE *__restrict__ Dx = (GB_B_TYPE *) B->x ;
    GB_C_TYPE *__restrict__ Cx = (GB_C_TYPE *) C->x ;

    #define A_iso GB_A_ISO
    #define D_iso GB_B_ISO

    const int64_t *__restrict__ Ai = A->i ;
    const int64_t *__restrict__ Ap = A->p ;
    GB_A_NVALS (anz) ; // this should be GB_A_NHELD for bitmap/full case
    const int64_t anvec = A->nvec ;

    // TODO: Handle bitmap/full case

    for (int64_t pfirst = blockIdx.x << log2_chunk_size ;
                 pfirst < anz ;
                 pfirst += gridDim.x << log2_chunk_size )
        {
            int64_t my_chunk_size, anvec_sub1 ;
            float slope ;
            int64_t kfirst = GB_cuda_ek_slice_setup (Ap, anvec, anz, pfirst,
                chunk_size, &my_chunk_size, &anvec_sub1, &slope) ;
            
            // alternate:
            // why not just do ek_slice_setup for one thread per block then this_thread_block.sync()?
            // answer:
            // better than having a syncrhonization barrier
            
            // question: why chunks are necessary? why not just do ek_slice_setup across all entries in one go?
            // answer: the slope method is only useful for a small range of entries; non-uniform entry distributions
            //         can distort the usefulness of the slope (will require an exhaustive linear search)
            //         for a large range of entries

            for (int64_t curr_p = threadIdx.x ; curr_p < my_chunk_size ; curr_p += blockDim.x)
            {
                int64_t k = GB_cuda_ek_slice_entry (curr_p, pfirst, Ap, anvec_sub1, kfirst, slope) ;

                GB_DECLAREB (dii) ;
                GB_GETB (dii, Dx, k, D_iso) ;
                GB_DECLAREA (aij) ;
                GB_GETA (aij, Ax, pfirst + curr_p, A_iso) ;
                GB_EWISEOP (Cx, pfirst + curr_p, aij, dii, 0, 0) ;
            }
        }
}

extern "C" {
    GB_JIT_CUDA_KERNEL_COLSCALE_PROTO (GB_jit_kernel) ;
}

GB_JIT_CUDA_KERNEL_COLSCALE_PROTO (GB_jit_kernel)
{
    ASSERT (GB_JUMBLED_OK (C)) ;
    ASSERT (GB_JUMBLED_OK (A)) ;
    ASSERT (!GB_JUMBLED (D)) ;
    ASSERT (!C->iso) ;

    dim3 grid (gridsz) ;
    dim3 block (blocksz) ;
    
    GB_cuda_colscale_kernel <<<grid, block, 0, stream>>> (C, A, D) ;

    return (GrB_SUCCESS) ;
}
